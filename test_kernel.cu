
#include <hip/hip_runtime.h>
// test_kernel.cu
extern __device__ float mathop(float & x, float & y, float & z);

__global__ void kernel(float *xvals, float * yvals, float * zvals, float *res)
{

        int tid = threadIdx.x + blockIdx.x * blockDim.x;
        res[tid] = mathop(xvals[tid], yvals[tid], zvals[tid]);
}

__global__ void kernel2(float *xvals, float * yvals, float * zvals, float *res)
{

       kernel<<<1,1>>>(xvals,yvals,zvals,res);
}